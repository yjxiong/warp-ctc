#include "hip/hip_runtime.h"
#include "helper.hpp"

__global__ void permute_kernel(const size_t* dims, size_t ndim, size_t n,
                          float* dst_data, size_t dst_dim, float beta,
                          const float* src_data, size_t src_dim, float alpha){
  for (int src_idx = blockIdx.x * blockDim.x + threadIdx.x; \
       src_idx < (n); \
       src_idx += blockDim.x * gridDim.x){

    size_t src_dim_idx = 0;
    size_t dst_dim_idx = 0;
    for (int i = 0, p = src_idx; i < ndim; ++i){
      size_t d = dims[i];
      if (i == src_dim) src_dim_idx = p % d;
      if (i == dst_dim) dst_dim_idx = p % d;
      p /= d;
    }

    size_t dst_idx = 0;
    for (int i = ndim - 1, p = src_idx, q = n; i >= 0; --i){

      size_t offset;
      size_t d;
      if (i == src_dim){
        d = dims[dst_dim]; q /= d;
        offset = dst_dim_idx;
      }else if(i == dst_dim){
        d = dims[src_dim]; q /= d;
        offset = src_dim_idx;
      }else{
        d = dims[i]; q /= d;
        offset = p / q;
      }

      dst_idx = dst_idx * d + offset;
      p %= q;
    }

    dst_data[dst_idx] = src_data[src_idx] * alpha + dst_data[dst_idx] * beta;
  }
}

#define MAX_THREAD 1024

void permute_dimension_gpu(const size_t* dims, size_t ndim, size_t num,
                           float* dst_data, size_t dst_dim, float beta,
                           const float* src_data, size_t src_dim, float alpha){

  if (src_dim == dst_dim) return;

  size_t num_threads = (num + MAX_THREAD - 1) / MAX_THREAD;
  permute_kernel  // NOLINT_NEXT_LINE(whitespace/operators)
  <<<num_threads, MAX_THREAD>>>(dims, ndim, num, dst_data, dst_dim, beta,
                          src_data, src_dim, alpha);

}